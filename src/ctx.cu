# include <ctx.h>

# include <hip/hip_runtime.h>
# include <cudaTypedefs.h>
# include <hip/hip_runtime.h>

using namespace FGPRS;

MyContext::MyContext(unsigned smCount)
{
	this->smCount = smCount;
	busy = false;
}

bool MyContext::initialize()
{
	CUexecAffinityParam_v1 affinity;
	_affinity.type = CU_EXEC_AFFINITY_TYPE_SM_COUNT;
	affinity.param.smCount.val = smCount;

	return cuCtxCreate_v3(&_context, &affinity, 1, 0, 0) == hipSuccess;
}

bool MyContext::select()
{
	if (busy)
		return false;
	
	busy = true;
	return hipCtxSetCurrent(_context) == hipSuccess;
}

bool MyContext::selectDefault()
{
	return hipCtxSetCurrent(0) == hipSuccess;
}

bool MyContext::release()
{
	busy = false;
	hipCtxSynchronize();
	return selectDefault();
}

bool MyContext::destroy()
{
	selectDefault();
	return hipCtxDestroy(_context) == hipSuccess;
}