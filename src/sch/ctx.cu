# include <ctx.h>

# include <schd.h>

# include <iostream>

# include <hip/hip_runtime.h>
# include <cudaTypedefs.h>
# include <hip/hip_runtime.h>

# include <torch/torch.h>

using namespace FGPRS;

using namespace std;
using namespace torch;

MyContext::MyContext()
{
	_default = true;
	this->smCount = Scheduler::maxSmCount;
	queueDuration = 0;
	_pMutex = new mutex();
}

MyContext::MyContext(unsigned smCount)
{
	_default = false;
	this->smCount = smCount;
	queueDuration = 0;
	_pMutex = new mutex();
}

bool MyContext::initialize()
{
	if (_default)
		return true;

	CUexecAffinityParam_v1 affinity;
	affinity.type = CU_EXEC_AFFINITY_TYPE_SM_COUNT;
	affinity.param.smCount.val = smCount;
	auto result = cuCtxCreate_v3(&_context, &affinity, 1, 0, 0);
	
	return result == hipSuccess;
}

bool MyContext::select(double duration)
{
	queueDuration += (unsigned long)(duration * 1000000);

	if (_default)
		return MyContext::selectDefault();
	
	// if (busy)
	// 	return false;

	busy = true;

	return hipCtxSetCurrent(_context) == hipSuccess;
}

bool MyContext::selectDefault()
{
	return hipCtxSetCurrent(0) == hipSuccess;
}

bool MyContext::release(double duration)
{
	queueDuration -= (unsigned long)(duration * 1000000);
	busy = false;
	// hipCtxSynchronize();
	// torch::cuda::synchronize();
	return selectDefault();
}

bool MyContext::destroy()
{
	selectDefault();
	
	if (_default)
		return true;
	
	return hipCtxDestroy(_context) == hipSuccess;
}

void MyContext::lock()
{
	_pMutex->lock();
}

void MyContext::unlock()
{
	_pMutex->unlock();
}